#include "hip/hip_runtime.h"
// Giorgio Gamba 11/2/2024
// Basic CUDA vector sum code using Unified Memory Access

// Cuda libraries
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

// Standard libraries
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

__global__ void vectorSumUM(int* a, int* b, int* c, int length)
{
	const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid < length)
		c[tid] = a[tid] + b[tid];
}

void initVector(int* v, int length)
{
	for (int i = 0; i < length; i++)
	{
		v[i] = rand() % 100;
	}
}

int main()
{
	int ID = hipGetDevice(&ID);

	int length = 1 << 16;

	size_t bytes = length * sizeof(int);

	// Unified memory pointers
	int* a = nullptr;
	int* b = nullptr;
	int* c = nullptr;

	initVector(a, length);
	initVector(b, length);

	// Using this "managed" version permits to automatically move from CPU to GPU and viceversa when needed
	hipMallocManaged(&a, bytes);
	hipMallocManaged(&b, bytes);
	hipMallocManaged(&c, bytes);

	const int BLOCK_SIZE = 256;								// Number of threads per block
	const int GRID_SIZE = (int)ceil(length / BLOCK_SIZE);	// Number of blocks per grid

	// Starts transferring information from CPU to GPU
	hipMemPrefetchAsync(a, bytes, ID);
	hipMemPrefetchAsync(b, bytes, ID);

	vectorSumUM<<<GRID_SIZE, BLOCK_SIZE >> > (a, b, c, length);

	// Waits for all the events of the GPU to be finished before continuing,
	// because CUDA operations are asynchronous. This is needed becase we use the Unified Memory Access.
	// In the other vector sum example, we didn't need it because the cudaMemCopy
	// was the synchronization barrier for all threads
	hipDeviceSynchronize();

	// Starts collecting information from GPU
	hipMemPrefetchAsync(c, bytes, hipCpuDeviceId);

	return 0;
}