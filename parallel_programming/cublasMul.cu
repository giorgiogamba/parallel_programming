// Copyright Giorgio Gamba
// Cublas multiplication testing

#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <hip/hip_runtime.h>


#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// NOTE: Cublas is assuming that the memory is made of contiguous COLUMNS inside the memory, 
// on the opposite of a CPU layout where there are contiguous ROWS

// For the float multiplication check we use an epsilon to check if we are close enough
// we are not looking for the exact multiplication

int cublasMul()
{
	int n = 1 << 10;
	size_t matrixSize = n * n * sizeof(float);

	// Vectors creation and memory allocation
	float *h_a, *h_b, *h_c;
	float *d_a, *d_b, *d_c;

	h_a = (float*)malloc(matrixSize);
	h_b = (float*)malloc(matrixSize);
	h_c = (float*)malloc(matrixSize);

	hipMalloc(&d_a, matrixSize);
	hipMalloc(&d_b, matrixSize);
	hipMalloc(&d_c, matrixSize);

	// Random values generation using cuda random generation framework
	hiprandGenerator_t randomGen;
	hiprandCreateGenerator(&randomGen, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Define the random generation using the system clock as seed
	hiprandSetPseudoRandomGeneratorSeed(randomGen, (unsigned long long)clock());

	// Defines the two input matrices in a random way using GPU randomization defining n*n elements
	// There's no need to make copies since we can create values and save them directly on GPU
	hiprandGenerateUniform(randomGen, d_a, n * n);
	hiprandGenerateUniform(randomGen, d_b, n * n);

	// Define multiplication operation
	hipblasHandle_t multiplicationHandle;
	hipblasCreate(&multiplicationHandle);

	// Parameters used for generalized multiplication
	// sgemm performs the following multiplication: c = (alpha * a) * b + (beta * c)
	// thus we cande fine alpha = 1 and beta + 0 to get a regular multiplication
	const float alpha = 1.0f;
	const float beta = 0.0f;
	hipblasSgemm(multiplicationHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_a, n, d_b, n, &beta, d_c, n);

	hipMemcpy(h_a, d_a, matrixSize, hipMemcpyDeviceToHost);
	hipMemcpy(h_b, d_b, matrixSize, hipMemcpyDeviceToHost);
	hipMemcpy(h_c, d_c, matrixSize, hipMemcpyDeviceToHost);

	// Solution verification can be done in the classic way, which is to assume that the matrix is composed by contiguous vectors,
	// but keeping in mind that they are placed by columns

	return 0;
}