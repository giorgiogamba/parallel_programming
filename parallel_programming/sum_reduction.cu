#include "hip/hip_runtime.h"
// Copyright Giorgio Gamba
// This algorithm is an example of a sum reduction algorithm with warp divergence

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <>

#define TBSIZE 256				// size of a single thread block
#define SHMEMSIZE TBSIZE * 4	// number of threads timers the integer size

// Basically we instantiate the shared memory to contains thread for each chunk of vector,
// but in the end we use only half of them at each iteration (respect to the previous one)

// @param v input
// @param r output
__global__ void my_sum_reduction(int* v, int* r)
{
	// Allocate shared memory
	__shared__ int partialSum[SHMEMSIZE];

	// Compute thread ID
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Load elements inside shared memory
	partialSum[threadIdx.x] = v[tid];
	__syncthreads();

	// Iteration based on a log base 2 block dimension
	for (int i = 1; i < blockDim.x; i *= 2)
	{
		// Given the pattern of the algorithm, we reduce the number of working threads at each iteration by 2

		// Each thread adds adjecents blocks together (e.g.: thread 0 adds block 0 and block 1)

		if (threadIdx.x % (2 * i) == 0)
			partialSum[threadIdx.x] += partialSum[threadIdx.x + i];

		__syncthreads();
	}

	// At the end on calculation, only the thread at position 0 will be active
	if (threadIdx.x == 0)
		r[blockIdx.x] = partialSum[0];
}

// This is an inefficient implementation that uses modulo op
__global__ void sun_reduction_diverged(int* input, int* output)
{
	__shared__ int partial_sum[SHMEMSIZE];

	int threadID = blockID.x * blockDim.x + threadIdx.x;

	// Loads elements inside shared memory
	// Each thread loads its input number inside its memory location inside the shared memory
	partial_sum[threadIdx.x] = input[threadID];
	__syncthreads();

	// Iteration in a log2 base manner the lock dimension, because at each step we are summing the element to the thread on the left
	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		// DIVERGED Implementation
	 
		// Takes only threads in even position and doubling always the distance -> log2 space
		if (threadIdx.x % (2 * stride) == 0)
		{
			partial_sum[threadIdx.x] += partial_sum[threadIdx.x + stride];
		}

		// BANK CONFLICT Implementation
		// The double faster
		// Computes index without using modulo operation
		// With this operation we see that the thread is shift to the right at each iteration -> only the lower index threads keep working
		int idx = 2 * stride * threadIdx.x;

		if (idx < blockDim.x)
		{
			partial_sum[idx] += partial_sum[idx + stride];
		}

		syncthreads();
	}

	// At the end on calculation, only the thread at position 0 will be active
	if (threadIdx.x == 0)
		output[blockIdx.x] = partial_sum[0];
}

int sum_reduction_test()
{
	// Initialize vector of 2^16 elements
	const int length = 1 << 16; // 2^16
	const size_t bytes = length * sizeof(int);

	// Create pointers for original vector and resulting vector, both on CPU and GPU
	int *hv, *hr;
	int* dv, * dr;

	// CPU Memory allocation
	hv = (int*)malloc(bytes);
	hr = (int*)malloc(bytes);

	// GPU Memory allocation
	hipMalloc(&dv, bytes);
	hipMalloc(&dr, bytes);

	// Random vector initialization
	for (size_t i = 0; i < length; ++i)
		hv[i] = rand() % 99;

	hipMemcpy(dv, hv, bytes, hipMemcpyHostToDevice);

	// Create 256 thread blocks of size 256, so that we can perform partial sum on each one of them
	// and then sum them together

	const int gridSize = (int)ceil(length / TBSIZE);

	// First, we computer inside 256 block 256 partial sums
	my_sum_reduction <<<gridSize, TBSIZE >>> (dv, dr);

	// Then, we will have a vector of 256 elements, each one representing a partial sum, and we reduce it
	// (note that we use the resulting vector as input also
	my_sum_reduction<<<1, TBSIZE>>>(dr, dr);

	hipMemcpy(hr, dr, bytes, hipMemcpyDeviceToHost);

	// Print the result
	printf("Result %d", hr[0]);

	return 0;
}